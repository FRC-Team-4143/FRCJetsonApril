#include "hip/hip_runtime.h"
#include "frc971/orin/threshold.h"

#include <stdint.h>

#include "frc971/orin/hip/hip_runtime.h"

namespace frc971::apriltag {
namespace {

// 1280 -> 2 * 128 * 5
// 720 -> 2 * 8 * 5 * 9
//
// 1456 -> 2 * 8 * 7 * 13
// 1088 -> 2 * 32 * 17

// Writes out the grayscale image and decimated image.
__global__ void InternalCudaToGreyscaleAndDecimateHalide(
    const uint8_t *color_image, uint8_t *gray_image, uint8_t *decimated_image,
    size_t width, size_t height) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < width * height) {
    uint8_t pixel = gray_image[i] = color_image[i * 2];

    const size_t row = i / width;
    const size_t col = i - width * row;

    // Copy over every other pixel.
    if (row % 2 == 0 && col % 2 == 0) {
      size_t decimated_row = row / 2;
      size_t decimated_col = col / 2;
      decimated_image[decimated_row * width / 2 + decimated_col] = pixel;
    }
    i += blockDim.x * gridDim.x;
  }

  // TODO(austin): Figure out how to load contiguous memory reasonably
  // efficiently and max/min over it.

  // TODO(austin): Can we do the threshold here too?  That would be less memory
  // bandwidth consumed...
}

// Returns the min and max for a row of 4 pixels.
__forceinline__ __device__ uchar2 minmax(uchar4 row) {
  uint8_t min_val = std::min(std::min(row.x, row.y), std::min(row.z, row.w));
  uint8_t max_val = std::max(std::max(row.x, row.y), std::max(row.z, row.w));
  return make_uchar2(min_val, max_val);
}

// Returns the min and max for a set of min and maxes.
__forceinline__ __device__ uchar2 minmax(uchar2 val0, uchar2 val1) {
  return make_uchar2(std::min(val0.x, val1.x), std::max(val0.y, val1.y));
}

// Returns the pixel index of a pixel at the provided x and y location.
__forceinline__ __device__ size_t XYToIndex(size_t width, size_t x, size_t y) {
  return width * y + x;
}

// Computes the min and max pixel value for each block of 4 pixels.
__global__ void InternalBlockMinMax(const uint8_t *decimated_image,
                                    uchar2 *unfiltered_minmax_image,
                                    size_t width, size_t height) {
  uchar2 vals[4];
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  for (int i = 0; i < 4; ++i) {
    const uchar4 decimated_block = *reinterpret_cast<const uchar4 *>(
        decimated_image + XYToIndex(width * 4, x * 4, y * 4 + i));

    vals[i] = minmax(decimated_block);
  }

  unfiltered_minmax_image[XYToIndex(width, x, y)] =
      minmax(minmax(vals[0], vals[1]), minmax(vals[2], vals[3]));
}

// Filters the min/max for the surrounding block of 9 pixels centered on our
// location using min/max and writes the result back out.
__global__ void InternalBlockFilter(const uchar2 *unfiltered_minmax_image,
                                    uchar2 *minmax_image, size_t width,
                                    size_t height) {
  uchar2 result = make_uchar2(255, 0);

  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  // Iterate through the 3x3 set of points centered on the point this image is
  // responsible for, and compute the overall min/max.
#pragma unroll
  for (int i = -1; i <= 1; ++i) {
#pragma unroll
    for (int j = -1; j <= 1; ++j) {
      const ssize_t read_x = x + i;
      const ssize_t read_y = y + j;

      if (read_x < 0 || read_x >= static_cast<ssize_t>(width)) {
        continue;
      }
      if (read_y < 0 || read_y >= static_cast<ssize_t>(height)) {
        continue;
      }

      result = minmax(
          result, unfiltered_minmax_image[XYToIndex(width, read_x, read_y)]);
    }
  }

  minmax_image[XYToIndex(width, x, y)] = result;
}

// Thresholds the image based on the filtered thresholds.
__global__ void InternalThreshold(const uint8_t *decimated_image,
                                  const uchar2 *minmax_image,
                                  uint8_t *thresholded_image, size_t width,
                                  size_t height, size_t min_white_black_diff) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < width * height) {
    const size_t x = i % width;
    const size_t y = i / width;

    const uchar2 minmax_val = minmax_image[x / 4 + (y / 4) * width / 4];

    uint8_t result;
    if (minmax_val.y - minmax_val.x < min_white_black_diff) {
      result = 127;
    } else {
      uint8_t thresh = minmax_val.x + (minmax_val.y - minmax_val.x) / 2;
      if (decimated_image[i] > thresh) {
        result = 255;
      } else {
        result = 0;
      }
    }

    thresholded_image[i] = result;
    i += blockDim.x * gridDim.x;
  }
}

}  // namespace

void CudaToGreyscaleAndDecimateHalide(
    const uint8_t *color_image, uint8_t *gray_image, uint8_t *decimated_image,
    uint8_t *unfiltered_minmax_image, uint8_t *minmax_image,
    uint8_t *thresholded_image, size_t width, size_t height,
    size_t min_white_black_diff, CudaStream *stream) {
  CHECK((width % 8) == 0);
  CHECK((height % 8) == 0);
  constexpr size_t kThreads = 256;
  {
    // Step one, convert to gray and decimate.
    size_t kBlocks = (width * height + kThreads - 1) / kThreads / 4;
    InternalCudaToGreyscaleAndDecimateHalide<<<kBlocks, kThreads, 0,
                                               stream->get()>>>(
        color_image, gray_image, decimated_image, width, height);
    MaybeCheckAndSynchronize();
  }

  size_t decimated_width = width / 2;
  size_t decimated_height = height / 2;

  {
    // Step 2, compute a min/max for each block of 4x4 (16) pixels.
    dim3 threads(16, 16, 1);
    dim3 blocks((decimated_width / 4 + 15) / 16,
                (decimated_height / 4 + 15) / 16, 1);

    InternalBlockMinMax<<<blocks, threads, 0, stream->get()>>>(
        decimated_image, reinterpret_cast<uchar2 *>(unfiltered_minmax_image),
        decimated_width / 4, decimated_height / 4);
    MaybeCheckAndSynchronize();

    // Step 3, Blur those min/max's a further +- 1 block in each direction using
    // min/max.
    InternalBlockFilter<<<blocks, threads, 0, stream->get()>>>(
        reinterpret_cast<uchar2 *>(unfiltered_minmax_image),
        reinterpret_cast<uchar2 *>(minmax_image), decimated_width / 4,
        decimated_height / 4);
    MaybeCheckAndSynchronize();
  }

  {
    // Now, write out 127 if the min/max are too close to each other, or 0/255
    // if the pixels are above or below the average of the min/max.
    size_t kBlocks = (width * height / 4 + kThreads - 1) / kThreads / 4;
    InternalThreshold<<<kBlocks, kThreads, 0, stream->get()>>>(
        decimated_image, reinterpret_cast<uchar2 *>(minmax_image),
        thresholded_image, decimated_width, decimated_height,
        min_white_black_diff);
    MaybeCheckAndSynchronize();
  }
}

}  // namespace frc971::apriltag
