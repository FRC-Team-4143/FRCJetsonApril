#include "hip/hip_runtime.h"
#include "frc971/orin/gpu_apriltag.h"

#include <chrono>

#include "absl/flags/flag.h"
#include "third_party/apriltag/apriltag.h"
#include "third_party/apriltag/apriltag_pose.h"
#include "third_party/apriltag/tag16h5.h"
#include "third_party/apriltag/tag36h11.h"
#include <opencv2/highgui.hpp>

//#include "aos/events/shm_event_loop.h" //RJS
//#include "aos/init.h"
//#include "aos/logging/logging.h"
#include "aos/realtime.h"
//#include "frc971/constants/constants_sender_lib.h"
#include "frc971/orin/apriltag.h"
//#include "frc971/vision/calibration_generated.h"
//#include "frc971/vision/charuco_lib.h"
//#include "frc971/vision/vision_util_lib.h"

ABSL_FLAG(bool, debug, false, "If true, write debug images.");
ABSL_FLAG(
    double, max_expected_distortion, 0.314,
    "Maximum expected value for unscaled distortion factors. Will scale "
    "distortion factors so that this value (and a higher distortion) maps to "
    "1.0.");
ABSL_FLAG(double, min_decision_margin, 50.0,
          "Minimum decision margin (confidence) for an apriltag detection");
ABSL_FLAG(int32_t, pixel_border, 150,
          "Size of image border within which to reject detected corners");
ABSL_FLAG(uint64_t, pose_estimation_iterations, 50,
          "Number of iterations for apriltag pose estimation.");

namespace frc971::apriltag {

// Set max age on image for processing at 20 ms.  For 60Hz, we should be
// processing at least every 16.7ms
constexpr aos::monotonic_clock::duration kMaxImageAge =
    std::chrono::milliseconds(50);

namespace chrono = std::chrono;

CameraMatrix GetCameraMatrix(
    const frc971::vision::calibration::CameraCalibration *calibration) {
  auto intrinsics = calibration->intrinsics();
  return CameraMatrix{
      .fx = intrinsics->Get(0),
      .cx = intrinsics->Get(2),
      .fy = intrinsics->Get(4),
      .cy = intrinsics->Get(5),
  };
}

DistCoeffs GetDistCoeffs(
    const frc971::vision::calibration::CameraCalibration *calibration) {
  auto dist_coeffs = calibration->dist_coeffs();
  return DistCoeffs{
      .k1 = dist_coeffs->Get(0),
      .k2 = dist_coeffs->Get(1),
      .p1 = dist_coeffs->Get(2),
      .p2 = dist_coeffs->Get(3),
      .k3 = dist_coeffs->Get(4),
  };
}

ApriltagDetector::ApriltagDetector(
    /* aos::EventLoop *event_loop, RJS */ /* std::string_view channel_name, RJS */
    const frc971::vision::calibration::CameraCalibration *calibration,
    size_t width, size_t height)
    : tag_family_(tag36h11_create()),
      tag_detector_(MakeTagDetector(tag_family_)),
      node_name_(event_loop->node()->name()->string_view()),
      calibration_(calibration),
      intrinsics_(frc971::vision::CameraIntrinsics(calibration_)),
      extrinsics_(frc971::vision::CameraExtrinsics(calibration_)),
      dist_coeffs_(frc971::vision::CameraDistCoeffs(calibration_)),
      distortion_camera_matrix_(GetCameraMatrix(calibration_)),
      distortion_coefficients_(GetDistCoeffs(calibration_)),
      gpu_detector_(width, height, tag_detector_, distortion_camera_matrix_,
                    distortion_coefficients_),
      /* image_callback_(
          event_loop, channel_name,
          [this](cv::Mat image_color_mat,
                 const aos::monotonic_clock::time_point eof) {
            HandleImage(image_color_mat, eof);
          },
          kMaxImageAge),
      target_map_sender_(
          event_loop->MakeSender<frc971::vision::TargetMap>(channel_name)),
      image_annotations_sender_(
          event_loop->MakeSender<foxglove::ImageAnnotations>(channel_name)), */
      rejections_(0) {
  //image_callback_.set_format(frc971::vision::ImageCallback::Format::YUYV2);

  projection_matrix_ = cv::Mat::zeros(3, 4, CV_64F);
  intrinsics_.rowRange(0, 3).colRange(0, 3).copyTo(
      projection_matrix_.rowRange(0, 3).colRange(0, 3));
}

ApriltagDetector::~ApriltagDetector() {
  apriltag_detector_destroy(tag_detector_);
  free(tag_family_);
}

apriltag_detector_t *ApriltagDetector::MakeTagDetector(
    apriltag_family_t *tag_family) {
  apriltag_detector_t *tag_detector = apriltag_detector_create();

  apriltag_detector_add_family_bits(tag_detector, tag_family, 1);

  tag_detector->nthreads = 6;
  tag_detector->wp = workerpool_create(tag_detector->nthreads);
  tag_detector->qtp.min_white_black_diff = 5;
  tag_detector->debug = absl::GetFlag(FLAGS_debug);

  return tag_detector;
}

flatbuffers::Offset<frc971::vision::TargetPoseFbs>
ApriltagDetector::BuildTargetPose(const Detection &detection,
                                  flatbuffers::FlatBufferBuilder *fbb) {
  const auto T =
      Eigen::Translation3d(detection.pose.t->data[0], detection.pose.t->data[1],
                           detection.pose.t->data[2]);
  const auto position_offset =
      frc971::vision::CreatePosition(*fbb, T.x(), T.y(), T.z());

  // Aprilrobotics stores the rotation matrix in row-major order
  const auto orientation = Eigen::Quaterniond(
      Eigen::Matrix<double, 3, 3, Eigen::RowMajor>(detection.pose.R->data));
  const auto orientation_offset = frc971::vision::CreateQuaternion(
      *fbb, orientation.w(), orientation.x(), orientation.y(), orientation.z());

  return frc971::vision::CreateTargetPoseFbs(
      *fbb, detection.det.id, position_offset, orientation_offset,
      detection.det.decision_margin, detection.pose_error,
      detection.distortion_factor, detection.pose_error_ratio);
}

bool ApriltagDetector::UndistortDetection(apriltag_detection_t *det) const {
  // Copy the undistorted points into det
  bool converged = true;
  for (size_t i = 0; i < 4; i++) {
    double u = det->p[i][0];
    double v = det->p[i][1];

    converged &= GpuDetector::UnDistort(&u, &v, &distortion_camera_matrix_,
                                        &distortion_coefficients_);
    det->p[i][0] = u;
    det->p[i][1] = v;
  }
  return converged;
}

double ApriltagDetector::ComputeDistortionFactor(
    const std::vector<cv::Point2f> &orig_corners,
    const std::vector<cv::Point2f> &corners) {
  CHECK_EQ(orig_corners.size(), 4ul);
  CHECK_EQ(corners.size(), 4ul);

  double avg_distance = 0.0;
  for (size_t i = 0; i < corners.size(); i++) {
    avg_distance += cv::norm(orig_corners[i] - corners[i]);
  }
  avg_distance /= corners.size();

  // Normalize avg_distance by dividing by the image diagonal,
  // and then the maximum expected distortion
  double distortion_factor =
      avg_distance /
      cv::norm(cv::Point2d(image_size_.width, image_size_.height));
  return std::min(
      distortion_factor / absl::GetFlag(FLAGS_max_expected_distortion), 1.0);
}

std::vector<cv::Point2f> ApriltagDetector::MakeCornerVector(
    const apriltag_detection_t *det) {
  std::vector<cv::Point2f> corner_points;
  corner_points.emplace_back(det->p[0][0], det->p[0][1]);
  corner_points.emplace_back(det->p[1][0], det->p[1][1]);
  corner_points.emplace_back(det->p[2][0], det->p[2][1]);
  corner_points.emplace_back(det->p[3][0], det->p[3][1]);

  return corner_points;
}

void ApriltagDetector::DestroyPose(apriltag_pose_t *pose) const {
  matd_destroy(pose->R);
  matd_destroy(pose->t);
}

void ApriltagDetector::HandleImage(cv::Mat color_image,
                                   aos::monotonic_clock::time_point eof) {
  const aos::monotonic_clock::time_point start_time =
      aos::monotonic_clock::now();
  gpu_detector_.Detect(color_image.data);
  image_size_ = color_image.size();
  cv::Mat image_copy;
  if (absl::GetFlag(FLAGS_visualize)) {
    // TODO: Need to figure out how to extract displayable color image from this
    image_copy = color_image.clone();
  }

  const zarray_t *detections = gpu_detector_.Detections();

  aos::monotonic_clock::time_point end_time = aos::monotonic_clock::now();

  const uint32_t min_x = absl::GetFlag(FLAGS_pixel_border);
  const uint32_t max_x = color_image.cols - absl::GetFlag(FLAGS_pixel_border);
  const uint32_t min_y = absl::GetFlag(FLAGS_pixel_border);
  const uint32_t max_y = color_image.rows - absl::GetFlag(FLAGS_pixel_border);

  // Define variables for storing / visualizing the output
  std::vector<Detection> results;
  auto builder = image_annotations_sender_.MakeBuilder();
  std::vector<flatbuffers::Offset<foxglove::PointsAnnotation>> foxglove_corners;

  for (int i = 0; i < zarray_size(detections); ++i) {
    apriltag_detection_t *gpu_detection;

    zarray_get(detections, i, &gpu_detection);

    bool valid = gpu_detection->decision_margin >
                 absl::GetFlag(FLAGS_min_decision_margin);

    if (valid) {
      // Reject tags that are too close to the boundary, since they often
      // lead to corrupt matches since part of the tag is cut off
      if (gpu_detection->p[0][0] < min_x || gpu_detection->p[0][0] > max_x ||
          gpu_detection->p[1][0] < min_x || gpu_detection->p[1][0] > max_x ||
          gpu_detection->p[2][0] < min_x || gpu_detection->p[2][0] > max_x ||
          gpu_detection->p[3][0] < min_x || gpu_detection->p[3][0] > max_x ||
          gpu_detection->p[0][1] < min_y || gpu_detection->p[0][1] > max_y ||
          gpu_detection->p[1][1] < min_y || gpu_detection->p[1][1] > max_y ||
          gpu_detection->p[2][1] < min_y || gpu_detection->p[2][1] > max_y ||
          gpu_detection->p[3][1] < min_y || gpu_detection->p[3][1] > max_y) {
        VLOG(1) << "Rejecting detection because corner is outside pixel border";

        // Send rejected corner points to foxglove in red
        std::vector<cv::Point2f> rejected_corner_points =
            MakeCornerVector(gpu_detection);
        foxglove_corners.push_back(frc971::vision::BuildPointsAnnotation(
            builder.fbb(), eof, rejected_corner_points,
            std::vector<double>{1.0, 0.0, 0.0, 0.5}));
        rejections_++;
        continue;
      }

      AOS_LOG(INFO,
              "Found GPU %s tag number %d hamming %d margin %f  (%f, %f), (%f, "
              "%f), (%f, %f), (%f, %f) in %f ms\n",
              valid ? "valid" : "invalid", gpu_detection->id,
              gpu_detection->hamming, gpu_detection->decision_margin,
              gpu_detection->p[0][0], gpu_detection->p[0][1],
              gpu_detection->p[1][0], gpu_detection->p[1][1],
              gpu_detection->p[2][0], gpu_detection->p[2][1],
              gpu_detection->p[3][0], gpu_detection->p[3][1],
              std::chrono::duration<float, std::milli>(end_time - start_time)
                  .count());

      VLOG(1) << "Found tag number " << gpu_detection->id
              << " hamming: " << gpu_detection->hamming
              << " margin: " << gpu_detection->decision_margin;

      // First create an apriltag_detection_info_t struct using your known
      // parameters.
      apriltag_detection_info_t info;
      info.tagsize = 6.5 * 0.0254;

      info.fx = intrinsics_.at<double>(0, 0);
      info.fy = intrinsics_.at<double>(1, 1);
      info.cx = intrinsics_.at<double>(0, 2);
      info.cy = intrinsics_.at<double>(1, 2);

      // Send original corner points in green
      std::vector<cv::Point2f> orig_corner_points =
          MakeCornerVector(gpu_detection);
      foxglove_corners.push_back(frc971::vision::BuildPointsAnnotation(
          builder.fbb(), eof, orig_corner_points,
          std::vector<double>{0.0, 1.0, 0.0, 0.5}));

      bool converged = UndistortDetection(gpu_detection);

      if (!converged) {
        VLOG(1) << "Rejecting detection because Undistort failed to coverge";

        // Send corner points rejected to to lack of convergence in orange
        std::vector<cv::Point2f> rejected_corner_points =
            MakeCornerVector(gpu_detection);
        foxglove_corners.push_back(frc971::vision::BuildPointsAnnotation(
            builder.fbb(), eof, rejected_corner_points,
            std::vector<double>{1.0, 0.65, 0.0, 0.5}));
        rejections_++;
        continue;
      }

      // We're setting this here to use the undistorted corner points in pose
      // estimation.
      info.det = gpu_detection;

      const aos::monotonic_clock::time_point before_pose_estimation =
          aos::monotonic_clock::now();

      apriltag_pose_t pose_1;
      apriltag_pose_t pose_2;
      double pose_error_1;
      double pose_error_2;
      estimate_tag_pose_orthogonal_iteration(
          &info, &pose_error_1, &pose_1, &pose_error_2, &pose_2,
          absl::GetFlag(FLAGS_pose_estimation_iterations));

      const aos::monotonic_clock::time_point after_pose_estimation =
          aos::monotonic_clock::now();
      VLOG(1) << "Took "
              << chrono::duration<double>(after_pose_estimation -
                                          before_pose_estimation)
                     .count()
              << " seconds for pose estimation";
      VLOG(1) << "Pose err 1: " << std::setprecision(20) << std::fixed
              << pose_error_1 << " " << (pose_error_1 < 1e-6 ? "Good" : "Bad");
      VLOG(1) << "Pose err 2: " << std::setprecision(20) << std::fixed
              << pose_error_2 << " " << (pose_error_2 < 1e-6 ? "Good" : "Bad");

      // Send undistorted corner points in pink
      std::vector<cv::Point2f> corner_points = MakeCornerVector(gpu_detection);
      foxglove_corners.push_back(frc971::vision::BuildPointsAnnotation(
          builder.fbb(), eof, corner_points,
          std::vector<double>{1.0, 0.75, 0.8, 1.0}));

      double distortion_factor =
          ComputeDistortionFactor(orig_corner_points, corner_points);

      // We get two estimates for poses.
      // Choose the one with the lower pose estimation error
      bool use_pose_1 = (pose_error_1 < pose_error_2);
      auto best_pose = (use_pose_1 ? pose_1 : pose_2);
      auto secondary_pose = (use_pose_1 ? pose_2 : pose_1);
      double best_pose_error = (use_pose_1 ? pose_error_1 : pose_error_2);
      double secondary_pose_error = (use_pose_1 ? pose_error_2 : pose_error_1);

      CHECK_NE(best_pose_error, std::numeric_limits<double>::infinity())
          << "Got no valid pose estimations, this should not be possible.";
      double pose_error_ratio = best_pose_error / secondary_pose_error;

      // Destroy the secondary pose if we got one
      if (secondary_pose_error != std::numeric_limits<double>::infinity()) {
        DestroyPose(&secondary_pose);
      }

      results.emplace_back(Detection{.det = *gpu_detection,
                                     .pose = best_pose,
                                     .pose_error = best_pose_error,
                                     .distortion_factor = distortion_factor,
                                     .pose_error_ratio = pose_error_ratio});

      if (absl::GetFlag(FLAGS_visualize)) {
        // Draw raw (distorted) corner points in green
        cv::line(image_copy, orig_corner_points[0], orig_corner_points[1],
                 cv::Scalar(0, 255, 0), 2);
        cv::line(image_copy, orig_corner_points[1], orig_corner_points[2],
                 cv::Scalar(0, 255, 0), 2);
        cv::line(image_copy, orig_corner_points[2], orig_corner_points[3],
                 cv::Scalar(0, 255, 0), 2);
        cv::line(image_copy, orig_corner_points[3], orig_corner_points[0],
                 cv::Scalar(0, 255, 0), 2);

        // Draw undistorted corner points in red
        cv::line(image_copy, corner_points[0], corner_points[1],
                 cv::Scalar(0, 0, 255), 2);
        cv::line(image_copy, corner_points[2], corner_points[1],
                 cv::Scalar(0, 0, 255), 2);
        cv::line(image_copy, corner_points[2], corner_points[3],
                 cv::Scalar(0, 0, 255), 2);
        cv::line(image_copy, corner_points[0], corner_points[3],
                 cv::Scalar(0, 0, 255), 2);
      }

      VLOG(1) << "Found tag number " << gpu_detection->id
              << " hamming: " << gpu_detection->hamming
              << " margin: " << gpu_detection->decision_margin;
    } else {
      rejections_++;
    }
  }

  if (absl::GetFlag(FLAGS_visualize)) {
    // Display the result
    // Rotate by 180 degrees to make it upright
    // TODO: Make this an option?
    bool flip_image_ = true;
    if (flip_image_) {
      cv::rotate(image_copy, image_copy, 1);
    }
    // TODO: Need to fix image display to handle YUYV images
    //    cv::imshow(absl::StrCat("ApriltagDetector Image ", node_name_),
    //               color_image);
    //    cv::waitKey(1);
  }

  const auto corners_offset = builder.fbb()->CreateVector(foxglove_corners);
  foxglove::ImageAnnotations::Builder annotation_builder(*builder.fbb());
  annotation_builder.add_points(corners_offset);
  builder.CheckOk(builder.Send(annotation_builder.Finish()));

  auto map_builder = target_map_sender_.MakeBuilder();
  std::vector<flatbuffers::Offset<frc971::vision::TargetPoseFbs>> target_poses;
  for (auto &detection : results) {
    auto *fbb = map_builder.fbb();
    auto pose = BuildTargetPose(detection, fbb);
    DestroyPose(&detection.pose);
    target_poses.emplace_back(pose);
  }
  const auto target_poses_offset =
      map_builder.fbb()->CreateVector(target_poses);
  auto target_map_builder =
      map_builder.MakeBuilder<frc971::vision::TargetMap>();
  target_map_builder.add_target_poses(target_poses_offset);
  target_map_builder.add_monotonic_timestamp_ns(eof.time_since_epoch().count());
  target_map_builder.add_rejections(rejections_);
  map_builder.CheckOk(map_builder.Send(target_map_builder.Finish()));

  // TODO: Do we need to clean this up?
  // apriltag_detections_destroy(detections);

  end_time = aos::monotonic_clock::now();

  if (absl::GetFlag(FLAGS_debug)) {
    timeprofile_display(tag_detector_->tp);
  }

  VLOG(2) << "Took " << chrono::duration<double>(end_time - start_time).count()
          << " seconds to detect overall";

  return;
  // TODO: Need to have proper return here
  //    return {.detections = results, .rejections = rejections_};
}

}  // namespace frc971::apriltag
